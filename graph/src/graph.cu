#include <iostream>
#include <gpugraph/graph.cuh>

namespace graph {
  graph::graph(size_t max_capacity) {
    // Initialize our graph memory into the GPU
    hipMalloc(&d_mem_.graph, max_capacity);

    // # Nodes will be the graph size
    hipMalloc(&d_mem_.nodes, max_capacity);

    // # Edges will be dynamic, but default to 3 * max_cap
    hipMalloc(&d_mem_.edges, 3 * max_capacity);
  }

  graph::~graph() {
    // Clean up our graph memory
    hipFree((void*)d_mem_.graph);
    hipFree((void*)d_mem_.nodes);
    hipFree((void*)d_mem_.edges);
  }
} // namespace graph
