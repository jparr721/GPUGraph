#include <iostream>
#include <gpugraph/graph.cuh>

namespace graph {
  graph::graph(size_t max_capacity) {
    // Initialize our graph memory into the GPU
    hipMalloc(&d_mem_.graph, max_capacity);

    // # Nodes will be the graph size
    hipMalloc(&d_mem_.nodes, max_capacity);

    // # Edges will be dynamic, but default to 3 * max_cap
    hipMalloc(&d_mem_.edges, 3 * max_capacity);
  }

  graph::~graph() {
    // Clean up our graph memory
    hipFree(d_mem_.graph);
    hipFree(d_mem_.nodes);
    hipFree(d_mem_.edges);
  }
} // namespace graph
